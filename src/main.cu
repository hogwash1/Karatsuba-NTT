#include "hip/hip_runtime.h"
#include <cstdlib>    // 标准库头文件
#include <random>     // 随机数生成库
#include <chrono>     // 时间库头文件
#include "ntt.cuh"    // GPU-NTT 库核心头文件
#include "lib/helper.cuh"

// #define DEFAULT_MODULUS  // 启用默认模数模式

using namespace std;
using namespace gpuntt;  // 使用 GPU-NTT 库的命名空间

// 全局参数声明
int LOGN;   // NTT变换的log2长度（实际长度N=2^LOGN）
int BATCH;  // 批量处理的多项式数量
int alpha;  
// 选择测试数据类型（64位版本）
typedef Data64 TestDataType;  // 定义用于测试的数据类型（32/64位可切换）

// 切分多项式
vector<vector<TestDataType>> split_poly(const vector<TestDataType> &a)
{
    int part_size = a.size() / 2; 
    vector<vector<TestDataType>> a_parts(2);
    for (int i = 0; i < 2; ++i)
    {
        a_parts[i] = vector<TestDataType>(a.begin() + i * part_size, a.begin() + (i + 1) * part_size);
    }
    return a_parts;
}

int main(int argc, char* argv[]) {
    // 初始化CUDA设备
    CudaDevice();  // 自定义CUDA设备初始化函数（可能包含错误检查）

    // 设置CUDA设备
    int device = 0;
    hipSetDevice(device);  // 选择0号GPU设备
    
    // 获取设备属性
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "最大网格尺寸: " 
            << prop.maxGridSize[0] << " x "
            << prop.maxGridSize[1] << " x " 
            << prop.maxGridSize[2] << std::endl;


    // 处理命令行参数
    if (argc < 3) {  // 默认参数
        LOGN = 12;    // 2^12 = 4096点NTT
        // BATCH = 1;    // 默认处理1个多项式
        alpha = 1;
    } else {          // 从命令行读取参数
        LOGN = atoi(argv[1]);  // 第一个参数为LOGN
        // BATCH = atoi(argv[2]); // 第二个参数为BATCH
        alpha = atoi(argv[2]);
    }
    // int alpha = 3;
    BATCH = 1 << alpha;

    cout << "LOGN = " << LOGN << ", BATCH = " << BATCH << endl;

    // 初始化NTT参数
#ifdef DEFAULT_MODULUS
    // 使用默认模数配置
    NTTParameters<TestDataType> parameters(LOGN, ReductionPolynomial::X_N_minus);
#else
    // 自定义模数配置
    NTTFactors<TestDataType> factor(Modulus<TestDataType>(576460752303415297), 288482366111684746, 238394956950829);
    NTTParameters<TestDataType> parameters(LOGN, factor, ReductionPolynomial::X_N_minus);
#endif

    // 创建CPU端NTT生成器
    // parameters.modulus.value = 268582913;
    NTTCPU<TestDataType> generator(parameters);  // 用于生成参考结果的CPU实现

    cout << "默认模数 = " << parameters.modulus.value << endl;
    // 初始化随机数生成器
    std::random_device rd;
    std::mt19937 gen(0);  // 固定种子用于结果可复现
    unsigned long long maxNumber = parameters.modulus.value - 1;
    // unsigned long long maxNumber = 5;
    std::uniform_int_distribution<unsigned long long> dis(0, maxNumber);

    cout << endl << "-----CPU 计算-----" << endl;
    // 生成随机输入数据
    cout << endl << "1.生成随机输入数据: input1, input2" << endl;
    vector<vector<TestDataType>> input1(BATCH), input2(BATCH);
    for (int j = 0; j < BATCH; j++) {
        for (int i = 0; i < parameters.n; i++) {  // parameters.n = 2^LOGN
            input1[j].push_back(dis(gen));  // 生成[0, modulus-1]范围内的随机数
            // input1[j].push_back(0);
            input2[j].push_back(dis(gen));
            // input2[j].push_back(2);
        }
        print_array(input1[j].data(), "input1[" + std::to_string(j) + "]");
        print_array(input2[j].data(), "input2[" + std::to_string(j) + "]");
    }
    // input1[0][0] = 1;input1[0][1] = 2;input1[0][parameters.n-1] = 3; input1[1][0] = 4;
    // input2[0][0] = 1;input2[0][1] = 1;input2[0][2] = 1;              input2[1][0] = 1;
    // input2[0][0] = 1;
    // 合并输入验证
    cout << endl << "合并输入验证 merge(input1), merge(input2)" << endl;
    vector<TestDataType> merged_input1, merged_input2;
    for (const auto &part : input1)
    {
        merged_input1.insert(merged_input1.end(), part.begin(), part.end());
    }
    // print_array(merged_input1.data(), "merged_input1");
    for(const auto &part : input2)
    {
        merged_input2.insert(merged_input2.end(), part.begin(), part.end());
    }
    // print_array(merged_input2.data(), "merged_input2");

    // NTT 乘法验证
    NTTParameters<TestDataType> parameters_merged(LOGN + alpha, ReductionPolynomial::X_N_minus);
    // parameters_merged.modulus.value = 268460033;
    // cout << "merged模数 = " << parameters_merged.modulus.value << endl;
    cout << endl << "INTT( NTT(merged_input1) * NTT(merged_input2) ) " << endl;

    // 添加计时器
    auto cpu_ntt_start = chrono::high_resolution_clock::now(); // 开始计时

    NTTCPU<TestDataType> generator_merged(parameters_merged);
    vector<TestDataType> merged_ntt_result1 = generator_merged.ntt(merged_input1);
    vector<TestDataType> merged_ntt_result2 = generator_merged.ntt(merged_input2); 
    vector<TestDataType> merged_ntt_result = generator_merged.mult(merged_ntt_result1, merged_ntt_result2);
    vector<TestDataType> intt_merged_ntt_result = generator_merged.intt(merged_ntt_result);
    print_array(intt_merged_ntt_result.data(), "intt_merged_ntt_result");

    auto cpu_ntt_end = chrono::high_resolution_clock::now(); // 结束计时
    chrono::duration<double, milli> cpu_ntt_duration = cpu_ntt_end - cpu_ntt_start;

    // cout << "merged_input1[n-1]:" << merged_input1[parameters.n-1] << endl;
    // 执行schoolbook乘法
    cout << endl << "schoolbook_poly_multiplication( merged_input1, merged_input2 )" << endl;

    // 添加计时器
    auto schoolbook_mult_start = chrono::high_resolution_clock::now(); // 开始计时
    vector<TestDataType> merged_schoolbook_result = schoolbook_poly_multiplication<TestDataType>(
    merged_input1, 
    merged_input2,
    parameters_merged.modulus,
    parameters_merged.poly_reduction  // 来自 NTTParameters
    );
    print_array(merged_schoolbook_result.data(), "merged_schoolbook_result");

    auto schoolbook_mult_end = chrono::high_resolution_clock::now(); // 结束计时
    chrono::duration<double, milli> schoolbook_mult_duration = schoolbook_mult_end - schoolbook_mult_start;
   


    // 给每个多项式补0
    for(int i = 0; i < BATCH; i++)
    {
        input1[i].resize(2*parameters.n, 0);
        input2[i].resize(2*parameters.n, 0);
    }
    NTTParameters<TestDataType> parameters_2n(LOGN+1, ReductionPolynomial::X_N_minus);
    NTTCPU<TestDataType> generator_2n(parameters_2n);

    // 执行CPU端NTT（生成参考结果）
    cout << endl << "2.CPU NTT: ntt(input1), ntt(input2) " << endl;
    vector<vector<TestDataType>> ntt_result(BATCH), ntt_result2(BATCH);
    for (int i = 0; i < BATCH; i++) {
        ntt_result[i] = generator_2n.ntt(input1[i]);  // CPU NTT计算
        ntt_result2[i] = generator_2n.ntt(input2[i]);
        print_array(ntt_result[i].data(), "ntt_result[" + std::to_string(i) + "]");
        print_array(ntt_result2[i].data(), "ntt_result2[" + std::to_string(i) + "]");
    }

    // Karatsuba 计算
    cout << endl << "3.Karatsuba(ntt(input1), ntt(input2)) " << endl;
    // 处理对角线项 diag_term[i] = ntt_result[i] * ntt_result2[i]
    vector<vector<TestDataType>> diag_term(BATCH);
    for(int i = 0; i < BATCH; i++)
    {
        diag_term[i].resize(2*parameters.n, 0);
        diag_term[i] = generator_2n.mult(ntt_result[i], ntt_result2[i]);
    }

    //  处理其它项 other_term[i] = ntt_result[i] * ntt_result2[i]
    vector<vector<TestDataType>> other_term(2 * BATCH, vector<TestDataType>(2*parameters.n, 0));
    for(int i = 0; i < BATCH; i++)
    {
        for(int j = i + 1; j < BATCH; j++)
        {
            // 计算fi*gj + fj*gi
            // 使用 (fi + fj) * (gi + gj) - fi*gi - fj*gj 优化计算
            vector<TestDataType> tmp1 = generator_2n.add(ntt_result[i], ntt_result[j]);
            vector<TestDataType> tmp2 = generator_2n.add(ntt_result2[i], ntt_result2[j]);
            vector<TestDataType> tmp3 = generator_2n.mult(tmp1, tmp2);
            tmp3 = generator_2n.sub(tmp3, diag_term[i]);
            tmp3 = generator_2n.sub(tmp3, diag_term[j]);
            other_term[i + j] = generator_2n.add(other_term[i + j], tmp3);
        }

    }
    // 对角线项加上其它项
    for (int i = 0; i < BATCH; ++i)
    {
        other_term[2 * i] = generator_2n.add(other_term[2 * i], diag_term[i]);
    }
    // other_term[i] + other_term[i + BATCH] 
    vector<vector<TestDataType>> ntt_result_final(BATCH);
    for(int i = 0; i < BATCH; ++i)
    {
        ntt_result_final[i].resize(2*parameters.n, 0);
        ntt_result_final[i] = generator_2n.add(other_term[i], other_term[i + BATCH]);
        print_array(ntt_result_final[i].data(), "ntt_result_final[" + std::to_string(i) + "]");
    }
    // INTT操作
    cout << endl << "4.INTT( Karatsuba( ntt(input1), ntt(input2) ) ) " << endl;
    vector<vector<TestDataType>> result_final(BATCH);
    for (int i = 0; i < BATCH; ++i)
    {
        result_final[i] = generator_2n.intt(ntt_result_final[i]);
        print_array(result_final[i].data(), "result_final[" + std::to_string(i) + "]");
    }
    // 合并结果
    vector<vector<vector<TestDataType>>> result_parts_split(BATCH);
    for (int i = 0; i < BATCH; ++i)
    {
        result_parts_split[i] = split_poly(result_final[i]); // 将每个部分拆分为两个基向量
    }
        
    for (int i = 0; i < BATCH; ++i)
    {
        // 前后相邻两个基向量相加 (第一个部分与最后一个部分相加)
        if (i == 0)
        {
            result_final[i] = generator_2n.add(result_parts_split[i][0], result_parts_split[BATCH - 1][1]);
            continue;
        }
        result_final[i] = generator_2n.add(result_parts_split[i - 1][1], result_parts_split[i][0]);
    }
    vector<TestDataType> result_merged;
    for (const auto &part : result_final)
    {
        result_merged.insert(result_merged.end(), part.begin(), part.end());
    }
    print_array(result_merged.data(), "result_merged");


   
    cout << endl << "LOGN = " << LOGN << ", " << "alpha = " << alpha << ", " 
    << "BATCH = " << BATCH << " , " << "LOGN + alpha = " << LOGN + alpha << endl;
    cout << "CPU NTT总耗时: " << cpu_ntt_duration.count() << " ms" << endl;
    cout << "Schoolbook multiplication 总耗时: " << schoolbook_mult_duration.count() << " ms" << endl;





    cout << endl << "-----------------GPU 计算-----------------" << endl;

    cout << endl << "-----------------普通 GPU-NTT 乘法-----------------" << endl;

    {
        // 在开始处添加总计时事件
        hipEvent_t total_start, total_stop;
        GPUNTT_CUDA_CHECK(hipEventCreate(&total_start));
        GPUNTT_CUDA_CHECK(hipEventCreate(&total_stop));
        // 计时
        hipEvent_t start, stop;
        GPUNTT_CUDA_CHECK(hipEventCreate(&start));
        GPUNTT_CUDA_CHECK(hipEventCreate(&stop));
        float elapsedTime,totalElapsedTime = 0;

        GPUNTT_CUDA_CHECK(hipEventRecord(total_start, 0));

        // 在主要计算开始前记录总耗时起点
        GPUNTT_CUDA_CHECK(hipEventRecord(total_start, 0));

        // 准备旋转因子表-----------------------------------------------------
        Root<TestDataType>* Forward_Omega_Table_Device;  // GPU端正向旋转因子表指针
        GPUNTT_CUDA_CHECK(
            hipMalloc(&Forward_Omega_Table_Device,
                    parameters_merged.root_of_unity_size * sizeof(Root<TestDataType>)));

        // 生成并拷贝旋转因子表
        vector<Root<TestDataType>> forward_omega_table = 
            parameters_merged.gpu_root_of_unity_table_generator(  // 生成旋转因子表
                parameters_merged.forward_root_of_unity_table);
        
        GPUNTT_CUDA_CHECK(hipMemcpy(Forward_Omega_Table_Device,
                                    forward_omega_table.data(),
                                    parameters_merged.root_of_unity_size * sizeof(Root<TestDataType>),
                                    hipMemcpyHostToDevice));
        
        // 准备逆旋转因子表-----------------------------------------------------
        Root<TestDataType>* Inverse_Omega_Table_Device;

        GPUNTT_CUDA_CHECK(
        hipMalloc(&Inverse_Omega_Table_Device,
                parameters_merged.root_of_unity_size * sizeof(Root<TestDataType>)));

        vector<Root<TestDataType>> inverse_omega_table =
        parameters_merged.gpu_root_of_unity_table_generator(
                    parameters_merged.inverse_root_of_unity_table);
        GPUNTT_CUDA_CHECK(hipMemcpy(Inverse_Omega_Table_Device,
                                    inverse_omega_table.data(),
                                    parameters_merged.root_of_unity_size * sizeof(Root<TestDataType>),
                                    hipMemcpyHostToDevice));


        // 配置模数参数-------------------------------------------------------
        Modulus<TestDataType>* test_modulus;  // GPU端模数参数指针
        GPUNTT_CUDA_CHECK(hipMalloc(&test_modulus, sizeof(Modulus<TestDataType>)));
        
        Modulus<TestDataType> test_modulus_[1] = {parameters_merged.modulus};  // 主机端模数
        GPUNTT_CUDA_CHECK(hipMemcpy(test_modulus, 
                                test_modulus_,
                                sizeof(Modulus<TestDataType>),
                                hipMemcpyHostToDevice));

        // 配置NTT参数--------------------------------------------------------
        ntt_rns_configuration<TestDataType> cfg_ntt = {
            .n_power = LOGN + alpha,                     // log2(N)
            .ntt_type = FORWARD,                 // 正向变换
            .reduction_poly = ReductionPolynomial::X_N_minus,  // 约减多项式类型
            .zero_padding = false,               // 无零填充
            .stream = 0                          // 使用默认流
        };

        // GPU内存分配与数据传输-----------------------------------------------
        TestDataType* Merge_Datas1, *Merge_Datas2;  // GPU输入/输出数据指针
        GPUNTT_CUDA_CHECK(  // 带错误检查的CUDA内存分配
                        hipMalloc(&Merge_Datas1,  parameters_merged.n * sizeof(TestDataType)));
        GPUNTT_CUDA_CHECK(  // 带错误检查的CUDA内存分配
                        hipMalloc(&Merge_Datas2,  parameters_merged.n * sizeof(TestDataType)));
        
        GPUNTT_CUDA_CHECK(
            hipMemcpy(Merge_Datas1,  // 目标地址
                       merged_input1.data(),                   // 源数据
                       parameters_merged.n * sizeof(TestDataType),// 数据大小
                       hipMemcpyHostToDevice));           // 传输方向
        GPUNTT_CUDA_CHECK(
            hipMemcpy(Merge_Datas2,  // 目标地址
                       merged_input2.data(),                   // 源数据
                       parameters_merged.n * sizeof(TestDataType),// 数据大小
                       hipMemcpyHostToDevice));           // 传输方向
                
        GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));
        // 执行GPU-NTT变换
        GPU_NTT_Inplace(  // 原地NTT变换
            Merge_Datas1,               // 输入/输出数据指针
            Forward_Omega_Table_Device,// 旋转因子表
            test_modulus,              // 模数参数
            cfg_ntt,                   // 配置参数
            1,                          // 批量数
            1                          // 流数量
        );
        GPU_NTT_Inplace(  // 原地NTT变换
            Merge_Datas2,               // 输入/输出数据指针
            Forward_Omega_Table_Device,// 旋转因子表
            test_modulus,              // 模数参数
            cfg_ntt,                   // 配置参数
            1,                          // 批量数
            1                          // 流数量
        );
        GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
        GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
        GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
        std::cout << "普通NTT耗时: " << elapsedTime << " ms" << std::endl;
        totalElapsedTime += elapsedTime;

        GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));
        // 执行乘法操作
        PointwiseMultiply(Merge_Datas1, Merge_Datas2, Merge_Datas1, parameters_merged.modulus, parameters_merged.n, 1);
        
        GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
        GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
        GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
        std::cout << "点乘耗时: " << elapsedTime << " ms" << std::endl;
        totalElapsedTime += elapsedTime;


        // 配置INTT模数参数-------------------------------------------------------
        Ninverse<TestDataType>* test_ninverse;
        GPUNTT_CUDA_CHECK(hipMalloc(&test_ninverse, sizeof(Ninverse<TestDataType>)));

        Ninverse<TestDataType> test_ninverse_[1] = {parameters_merged.n_inv};

        GPUNTT_CUDA_CHECK(hipMemcpy(test_ninverse, test_ninverse_,
                                    sizeof(Ninverse<TestDataType>), hipMemcpyHostToDevice));

        // 配置INTT参数--------------------------------------------------------
        ntt_rns_configuration<TestDataType> cfg_intt = {
            .n_power = LOGN + alpha,
            .ntt_type = INVERSE,
            .reduction_poly = ReductionPolynomial::X_N_minus,
            .zero_padding = false,
            .mod_inverse = test_ninverse,
            .stream = 0
        };


        GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));
        // 执行INTT变换
        GPU_NTT_Inplace(  // 原地NTT变换
            Merge_Datas1,               // 输入/输出数据指针
            Inverse_Omega_Table_Device,// 旋转因子表
            test_modulus,              // 模数参数
            cfg_intt,                   // 配置参数
            1,                          // 批量数
            1                          // 流数量
        );

        GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
        GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
        GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
        std::cout << "普通INTT耗时: " << elapsedTime << " ms" << std::endl;
        totalElapsedTime += elapsedTime;
        //  GPU_NTT(ntt_result_device, result_device, Inverse_Omega_Table_Device, test_modulus, cfg_intt, BATCH, 1);

        // 添加总耗时计算
        GPUNTT_CUDA_CHECK(hipEventRecord(total_stop, 0)); 
        GPUNTT_CUDA_CHECK(hipEventSynchronize(total_stop));
        float totalElapsed = 0;
        GPUNTT_CUDA_CHECK(hipEventElapsedTime(&totalElapsed, total_start, total_stop));
        std::cout << "\n----- 普通GPU-NTT总耗时分析 -----" << std::endl;
        std::cout << "[TOTAL] 核心计算耗时: " << totalElapsedTime << " ms" << std::endl;
        std::cout << "[TOTAL] 总耗时: " << totalElapsed << " ms" << std::endl;
        std::cout << "------------------------------" << std::endl;

        // 销毁总计时事件
        GPUNTT_CUDA_CHECK(hipEventDestroy(total_start));
        GPUNTT_CUDA_CHECK(hipEventDestroy(total_stop));

        // 销毁耗时统计
        GPUNTT_CUDA_CHECK(hipEventDestroy(start));
        GPUNTT_CUDA_CHECK(hipEventDestroy(stop));

        GPUNTT_CUDA_CHECK( hipFree(Merge_Datas1) );
        GPUNTT_CUDA_CHECK( hipFree(Merge_Datas2) );
        GPUNTT_CUDA_CHECK( hipFree(Forward_Omega_Table_Device) );
        GPUNTT_CUDA_CHECK( hipFree(Inverse_Omega_Table_Device) );
        GPUNTT_CUDA_CHECK( hipFree(test_modulus) );
        GPUNTT_CUDA_CHECK( hipFree(test_ninverse) );
    }


    cout << endl << "---------------Karatsuba-NTT乘法---------------" << endl;
    // 在GPU计算开始处添加总计时事件
    hipEvent_t total_start, total_stop;
    GPUNTT_CUDA_CHECK(hipEventCreate(&total_start));
    GPUNTT_CUDA_CHECK(hipEventCreate(&total_stop));
    // 计时
    hipEvent_t start, stop;
    GPUNTT_CUDA_CHECK(hipEventCreate(&start));
    GPUNTT_CUDA_CHECK(hipEventCreate(&stop));
    float elapsedTime,totalElapsedTime = 0;


    
    // 在主要计算开始前记录总耗时起点
    GPUNTT_CUDA_CHECK(hipEventRecord(total_start, 0));

    // 准备旋转因子表-----------------------------------------------------
    Root<TestDataType>* Forward_Omega_Table_Device;  // GPU端正向旋转因子表指针
    GPUNTT_CUDA_CHECK(
        hipMalloc(&Forward_Omega_Table_Device,
                parameters_2n.root_of_unity_size * sizeof(Root<TestDataType>)));

    // 生成并拷贝旋转因子表
    vector<Root<TestDataType>> forward_omega_table = 
        parameters_2n.gpu_root_of_unity_table_generator(  // 生成旋转因子表
            parameters_2n.forward_root_of_unity_table);
    
    GPUNTT_CUDA_CHECK(hipMemcpy(Forward_Omega_Table_Device,
                                forward_omega_table.data(),
                                parameters_2n.root_of_unity_size * sizeof(Root<TestDataType>),
                                hipMemcpyHostToDevice));

   // 准备逆旋转因子表-----------------------------------------------------
    Root<TestDataType>* Inverse_Omega_Table_Device;

    GPUNTT_CUDA_CHECK(
    hipMalloc(&Inverse_Omega_Table_Device,
            parameters_2n.root_of_unity_size * sizeof(Root<TestDataType>)));

    vector<Root<TestDataType>> inverse_omega_table =
    parameters_2n.gpu_root_of_unity_table_generator(
                parameters_2n.inverse_root_of_unity_table);
    GPUNTT_CUDA_CHECK(hipMemcpy(Inverse_Omega_Table_Device,
                                inverse_omega_table.data(),
                                parameters_2n.root_of_unity_size * sizeof(Root<TestDataType>),
                                hipMemcpyHostToDevice));


    // 配置模数参数-------------------------------------------------------
    Modulus<TestDataType>* test_modulus;  // GPU端模数参数指针
    GPUNTT_CUDA_CHECK(hipMalloc(&test_modulus, sizeof(Modulus<TestDataType>)));
    
    Modulus<TestDataType> test_modulus_[1] = {parameters_2n.modulus};  // 主机端模数
    GPUNTT_CUDA_CHECK(hipMemcpy(test_modulus, 
                               test_modulus_,
                               sizeof(Modulus<TestDataType>),
                               hipMemcpyHostToDevice));

    // 配置NTT参数--------------------------------------------------------
    ntt_rns_configuration<TestDataType> cfg_ntt = {
        .n_power = LOGN + 1,                     // log2(N)
        .ntt_type = FORWARD,                 // 正向变换
        .reduction_poly = ReductionPolynomial::X_N_minus,  // 约减多项式类型
        .zero_padding = false,               // 无零填充
        .stream = 0                          // 使用默认流
    };

    // GPU内存分配与数据传输-----------------------------------------------
    TestDataType* InOut_Datas, *InOut_Datas2;  // GPU输入/输出数据指针
    GPUNTT_CUDA_CHECK(  // 带错误检查的CUDA内存分配
        hipMalloc(&InOut_Datas, BATCH * parameters_2n.n * sizeof(TestDataType)));
    GPUNTT_CUDA_CHECK(  // 带错误检查的CUDA内存分配
        hipMalloc(&InOut_Datas2, BATCH * parameters_2n.n * sizeof(TestDataType)));
    
    // 分批拷贝数据到GPU
    for (int j = 0; j < BATCH; j++) {
        GPUNTT_CUDA_CHECK(
            hipMemcpy(InOut_Datas + (parameters_2n.n * j),  // 目标地址
                       input1[j].data(),                   // 源数据
                       parameters_2n.n * sizeof(TestDataType),// 数据大小
                       hipMemcpyHostToDevice));           // 传输方向
        GPUNTT_CUDA_CHECK(
            hipMemcpy(InOut_Datas2 + (parameters_2n.n * j) ,  // 目标地址
                       input2[j].data(),                   // 源数据
                       parameters_2n.n * sizeof(TestDataType),// 数据大小
                       hipMemcpyHostToDevice));           
    }

    // 执行GPU NTT--------------------------------------------------------

    // 测量NTT时间
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));

    GPU_NTT_Inplace(  // 原地NTT变换
        InOut_Datas,               // 输入/输出数据指针
        Forward_Omega_Table_Device,// 旋转因子表
        test_modulus,              // 模数参数
        cfg_ntt,                   // 配置参数
        BATCH,                     // 批量数
        1                          // 流数量
    );
    GPU_NTT_Inplace(  // 原地NTT变换
        InOut_Datas2,               // 输入/输出数据指针
        Forward_Omega_Table_Device,// 旋转因子表
        test_modulus,              // 模数参数
        cfg_ntt,                   // 配置参数
        BATCH,                     // 批量数
        1                          // 流数量
    );

    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "双路NTT总耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;



    TestDataType* diag_term_device; // GPU端对角项指针
    GPUNTT_CUDA_CHECK(hipMalloc(&diag_term_device, BATCH * parameters_2n.n * sizeof(TestDataType)));
    TestDataType* other_term_device; // GPU端非对角项指针
    GPUNTT_CUDA_CHECK(hipMalloc(&other_term_device, 2 * BATCH * parameters_2n.n * sizeof(TestDataType)));
    TestDataType* ntt_result_device; // GPU端NTT结果指针
    GPUNTT_CUDA_CHECK(hipMalloc(&ntt_result_device, BATCH * parameters_2n.n * sizeof(TestDataType)));
    GPUNTT_CUDA_CHECK(hipMemset(ntt_result_device, 0, BATCH * parameters_2n.n * sizeof(TestDataType)));
    TestDataType* result_device; // GPU端结果指针
    GPUNTT_CUDA_CHECK(hipMalloc(&result_device, BATCH * parameters_2n.n * sizeof(TestDataType))); 
    TestDataType* result_merged_device; // GPU端结果内部折叠指针
    GPUNTT_CUDA_CHECK(hipMalloc(&result_merged_device, BATCH * parameters.n * sizeof(TestDataType)));

    // TestDataType* diag_term_device;
    // GPUNTT_CUDA_CHECK(hipMalloc(&diag_term_device, BATCH * parameters_2n.n * sizeof(TestDataType)));

    // 处理对角线项 (i = j) diagonal term
    // 在点乘操作添加计时
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));

        PointwiseMultiply<TestDataType>(
            InOut_Datas,                      // 第一个多项式地址
            InOut_Datas2,                     // 第二个多项式地址
            diag_term_device,                 // 输出地址
            parameters_2n.modulus,               // 模数参数
            BATCH * parameters_2n.n,             // 多项式长度
            1,                                // 批量数
            0                                 // 使用默认流
        );

    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "对角线项计算耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;

    //验证
    TestDataType* Output_Host =  // 主机端结果缓冲区
    (TestDataType*)malloc(BATCH * parameters_2n.n * sizeof(TestDataType));
    GPUNTT_CUDA_CHECK(hipMemcpy(Output_Host,   // 目标地址
                               diag_term_device,   // 源数据
                               BATCH * parameters_2n.n * sizeof(TestDataType),
                               hipMemcpyDeviceToHost));
    // print_array(Output_Host, "GPU对角线项结果");
    VERIFY_RESULTS(Output_Host, diag_term, "GPU计算对角线项结果正确");

    // 处理其它项
    // 在其它项计算添加计时
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));

    karatsuba(InOut_Datas, InOut_Datas2, diag_term_device, other_term_device, parameters_2n.modulus, parameters_2n.n, BATCH);
    merge_diag_non_diag(diag_term_device, other_term_device, other_term_device, parameters_2n.modulus, parameters_2n.n, BATCH);
    
    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "非对角项计算耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;

    // GPUNTT_CUDA_CHECK(hipGetLastError());
    


    // 验证other_term计算结果
    TestDataType* host_other_term = (TestDataType*)malloc(2*BATCH*parameters_2n.n*sizeof(TestDataType));
    GPUNTT_CUDA_CHECK(hipMemcpy(host_other_term, other_term_device, 
                            2*BATCH*parameters_2n.n*sizeof(TestDataType),
                            hipMemcpyDeviceToHost));
    VERIFY_RESULTS(host_other_term, other_term, "GPU计算其它项结果正确");

    // 合并结果
    // 在结果合并添加计时
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));
    PointwiseAdd(other_term_device, other_term_device + (BATCH * parameters_2n.n), ntt_result_device, parameters_2n.modulus, BATCH * parameters_2n.n);

    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "合并对角线项和其它项耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;

    // 验证
    TestDataType* host_ntt_result = (TestDataType*)malloc(BATCH*parameters_2n.n*sizeof(TestDataType));
    GPUNTT_CUDA_CHECK(hipMemcpy(host_ntt_result, ntt_result_device, 
                            BATCH*parameters_2n.n*sizeof(TestDataType),
                            hipMemcpyDeviceToHost));
    // print_array(host_ntt_result, "GPU 计算合并结果");
    VERIFY_RESULTS(host_ntt_result, ntt_result_final, "GPU计算合并对角线项和其它项结果正确");

    // 配置INTT模数参数-------------------------------------------------------
    Ninverse<TestDataType>* test_ninverse;
    GPUNTT_CUDA_CHECK(hipMalloc(&test_ninverse, sizeof(Ninverse<TestDataType>)));

    Ninverse<TestDataType> test_ninverse_[1] = {parameters_2n.n_inv};

    GPUNTT_CUDA_CHECK(hipMemcpy(test_ninverse, test_ninverse_,
                                 sizeof(Ninverse<TestDataType>), hipMemcpyHostToDevice));

    // 配置INTT参数--------------------------------------------------------
    ntt_rns_configuration<TestDataType> cfg_intt = {
        .n_power = LOGN + 1,
        .ntt_type = INVERSE,
        .reduction_poly = ReductionPolynomial::X_N_minus,
        .zero_padding = false,
        .mod_inverse = test_ninverse,
        .stream = 0
    };
    
    // 执行GPU_INTT( ntt_result_device )--------------------------------------------------------
    // 在INTT操作添加计时
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));  

    // TestDataType* result_device;
    // GPUNTT_CUDA_CHECK(hipMalloc(&result_device, BATCH * parameters_2n.n * sizeof(TestDataType))); 
    GPU_NTT(ntt_result_device, result_device, Inverse_Omega_Table_Device, test_modulus, cfg_intt, BATCH, 1);

    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "INTT计算耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;
        
    // 合并结果
    // 在结果合并添加计时
    GPUNTT_CUDA_CHECK(hipEventRecord(start, 0));
    // TestDataType* result_merged_device;
    // GPUNTT_CUDA_CHECK(hipMalloc(&result_merged_device, BATCH * parameters.n * sizeof(TestDataType)));
    // #pragma unroll
    // for(int i = 0; i < BATCH; i++)
    // {
    //     TestDataType* result_i_0 = result_device + i * parameters_2n.n;                     // 表示 result[i][0]
    //     TestDataType* result_i_1 = result_device + ((i == 0) ? 
    //         (BATCH-1)*parameters_2n.n :             // 表示 result[BATCH-1][1]
    //         (i-1)*parameters_2n.n) + parameters.n;  // 表示 result[i-1][1]
    //     TestDataType* result_merged_i = result_merged_device + i * parameters.n;            // 表示 result_merged[i]
    //     // result_merged[i] = result[i][0] + result[i-1][1]
    //     PointwiseAdd(result_i_0 , result_i_1, result_merged_i, parameters_2n.modulus, parameters.n);    //使用parameters_2n.modulus
    // }
    merge(result_device, result_merged_device, parameters_2n.modulus, parameters.n, BATCH);

    


    GPUNTT_CUDA_CHECK(hipEventRecord(stop, 0));
    GPUNTT_CUDA_CHECK(hipEventSynchronize(stop));
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "合并结果耗时: " << elapsedTime << " ms" << std::endl;
    totalElapsedTime += elapsedTime;

    // 添加总耗时计算
    GPUNTT_CUDA_CHECK(hipEventRecord(total_stop, 0)); 
    GPUNTT_CUDA_CHECK(hipEventSynchronize(total_stop));
    float totalElapsed = 0;
    GPUNTT_CUDA_CHECK(hipEventElapsedTime(&totalElapsed, total_start, total_stop));
    std::cout << "\n----- 总耗时分析 -----" << std::endl;
    std::cout << "[TOTAL] 核心计算耗时: " << totalElapsedTime << " ms" << std::endl;
    std::cout << "[TOTAL] 总耗时: " << totalElapsed << " ms" << std::endl;
    std::cout << "------------------------------" << std::endl;

    // 销毁总计时事件
    GPUNTT_CUDA_CHECK(hipEventDestroy(total_start));
    GPUNTT_CUDA_CHECK(hipEventDestroy(total_stop));

    // 销毁耗时统计
    GPUNTT_CUDA_CHECK(hipEventDestroy(start));
    GPUNTT_CUDA_CHECK(hipEventDestroy(stop));

    // 验证
    TestDataType* host_result = (TestDataType*)malloc(BATCH * parameters.n * sizeof(TestDataType));
    GPUNTT_CUDA_CHECK(hipMemcpy(host_result,   // 目标地址
                            result_merged_device,   // 源数据
                            BATCH * parameters.n * sizeof(TestDataType),
                            hipMemcpyDeviceToHost));
    bool check = true; 
    for (int i = 0; i < BATCH; i++) 
    { 
        check = check_result( 
            host_result + (i * parameters.n), 
            result_final[i].data(), 
            parameters.n 
        ); 
        if (!check) { 
            std::cout << "第 " << i << " 个多项式验证失败" << std::endl; 
            break; 
        } 
    } 
    if (check) std::cout << "Karatsuba_GPU计算结果正确" << std::endl;
    // VERIFY_RESULTS(host_result, result_final, "Karatsuba_GPU计算结果正确");


    // 资源释放-----------------------------------------------------------
    GPUNTT_CUDA_CHECK(hipFree(InOut_Datas));
    GPUNTT_CUDA_CHECK(hipFree(InOut_Datas2));

    GPUNTT_CUDA_CHECK(hipFree(Forward_Omega_Table_Device));
    GPUNTT_CUDA_CHECK(hipFree(Inverse_Omega_Table_Device));
    free(Output_Host);

    GPUNTT_CUDA_CHECK(hipFree(diag_term_device));
    GPUNTT_CUDA_CHECK(hipFree(other_term_device));
    GPUNTT_CUDA_CHECK(hipFree(result_merged_device));

    return EXIT_SUCCESS;
}
// cmake --build .